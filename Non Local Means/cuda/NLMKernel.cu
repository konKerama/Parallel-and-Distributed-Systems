
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// Array access macros
#define INPUT(i,j) imgBef[(i)*n + j]
#define OUTPUT(i,j) imgAfter[(i)*n + j]
#define fNi(i,j) fNi[(i)*patchSize + j]
#define fNj(i,j) fNj[(i)*patchSize + j]
#define fN(i,j) fN[(i)*patchSize + j]
#define H(i,j) H[(i)*patchSize + j]
#define PATCH(i,j) patch[(i)*blockDim.y + j]
#define OTHER_PATCH(i,j) other_patch[(i)*blockDim.y + j]

/* If out of block -> get fNi from global memory
   Else get from patch (shared)
   Out of block happens when: k or l <= padSize  OR
   if k > blockDim.x + padSize or if l > blockDim.y + padSize
*/

__device__ int isInBlock(int k, int l, int padSize, int a, int b){

	if (k<0 || l<0 || k>a || l>b){
		return 0;
	}
	else {
		return 1;
	}
}

/* Calculates the norm between two vectors based on the gaussian matrix H for
   a predetermined patch size.
   */
__device__ float calcNorm(float *fNi, float *fNj, float *H, int patchSize){
	float sum =0;
	for (int k=0; k< patchSize; k++){
		for (int l=0; l<patchSize; l++){
			sum+=(fNi(k,l)-fNj(k,l))*(fNi(k,l)-fNj(k,l))*H(k,l);
		}
	}
	return sum;
}

/* 
 Checks if targetted pixel at the edges of the final blocks is outside of the image.
*/
__device__ int pixelOutofImg(int x, int y, int m, int n){

	if (x > m || y > n){
		return 1;
	}
	else{
		return 0;
	}

}

__device__ void computeFN(float * fN, int i, int j, int n, float * patch, float const * imgBef, int padSize, int patchSize, int currentBlock_X, int currentBlock_Y, int blockDimX, int blockDimY ){
	
	int a,b,k,l,e=0,f=0;
	for (k=i-padSize; k<=i+padSize; k++){
				for (l = j-padSize; l<=j+padSize; j++){ 
					a = k - currentBlock_X * blockDimX; // a e [0,patchSize)
					b = l - currentBlock_Y * blockDimY; // b e [0,patchSize)

					if (isInBlock(a, b, padSize, blockDimX, blockDimY)){
						fN(e,f) = PATCH(a,b);
					}
					else{
						fN(e,f) = INPUT(k,l);
					}
					f++;
					if (f==patchSize){
						f=0;
						e++;
					}
				}
			}	

}

__global__ void cudaNonLocalMeans(float const *imgBef, float *imgAfter, float *H, float filtSigma,
                                    int m, int n, int padSize) {
	// Get pixel (x,y) in input

	__shared__ extern float patch[];
	float *other_patch = patch + blockDim.x*blockDim.y;
	int currentBlock_X = blockIdx.x; // [0,numBlocksX)
	int currentBlock_Y = blockIdx.y; // [0,numBlocksY)

	int patchSize = 2*padSize+1;
	int notInPaddedArea=1;
	//float *other_patch = patch +
	int i = currentBlock_X * blockDim.x + threadIdx.x;
	int j = currentBlock_Y * blockDim.y + threadIdx.y;
	// nBlocks_X,Y is the number of blocks per axis
	int nBlocks_X = gridDim.x;
	int nBlocks_Y = gridDim.y;

	float Z=0;
	float exponent=0;
	float pixel=0;
	float *fNi, *fNj;

	// if i or j less than 2 or greater than 65 they are outside of the image.
	if (i < padSize || j < padSize || i > m-padSize-1 || j > n-padSize-1) notInPaddedArea=0;

	if (i < m && j < n){ // INSIDE IMAGE
		if (notInPaddedArea){
			PATCH(threadIdx.x, threadIdx.y) = INPUT(i,j);
			// fNi calculation
			computeFN(fNi, i, j, n, patch,imgBef,padSize,patchSize,currentBlock_X, currentBlock_Y, blockDim.x, blockDim.y);
			// fNj calculation for CURRENT block only.
			for (int row = currentBlock_X*blockDim.x ; row<(currentBlock_X+1)*blockDim.x; row++){
				for (int col = currentBlock_Y*blockDim.y ; col<(currentBlock_Y+1)*blockDim.y; col++){
					//if (row > m-padSize || col > n-padSize || row < padSize || col < padSize )continue;
					computeFN(fNj, row, col, n, patch,imgBef,padSize,patchSize,currentBlock_X, currentBlock_Y, blockDim.x, blockDim.y);
					exponent = calcNorm(fNi, fNj, H, patchSize);
					exponent/=filtSigma;
					Z+=exp(-exponent);
					pixel += exp(-exponent)*INPUT(row,col);
				}
			}
		}
	}
	__syncthreads();

	/*
	  So far the pixel has been affected only by the values of its own block.
	  The following code implements the effect of the rest of the blocks on
	  the pixel.
	*/

	int blockNoX, blockNoY;
	// For each block other than the current 
	for (blockNoX = 0; blockNoX < nBlocks_X ; blockNoX++){
		for (blockNoY = 0; blockNoY < nBlocks_Y ; blockNoY++){
			if (blockNoX!=currentBlock_X && blockNoY!=currentBlock_Y){
				// Save block to shared
				// For each pixel of the block copy to shared (other_patch)
				if (!(pixelOutofImg(blockNoX * blockDim.x + threadIdx.x, blockNoY * blockDim.y + threadIdx.y, m, n))){
					OTHER_PATCH(threadIdx.x,threadIdx.y)=INPUT(blockNoX * blockDim.x + threadIdx.x , blockNoY * blockDim.y + threadIdx.y);
				}
				__syncthreads();
				// If the pixel is inside the current block compute from shared, otherwise from global. 
				if (notInPaddedArea){
					for (int row = blockNoX*blockDim.x ; row<(blockNoX+1)*blockDim.x; row++){
						for (int col = blockNoY*blockDim.y ; col<(blockNoY+1)*blockDim.y; col++){
							if (row > m-padSize || col > n-padSize) continue;
							computeFN(fNj, row, col, n, other_patch,imgBef,padSize,patchSize,blockNoX,blockNoY, blockDim.x, blockDim.y);
							exponent = calcNorm(fNi, fNj, H, patchSize);
							exponent/=filtSigma;
							Z+=exp(-exponent);
							pixel += exp(-exponent)*INPUT(row,col);

						}
					}
				}
			}
		}
	}
	/* Finally since the total effect of the other pixels on the current pixel
	   has been calculated, pixel is divided by the finalized Z matrix ending up with its
	   final denoised version.
	*/	
	if (notInPaddedArea){
		OUTPUT(i,j) = pixel/Z;	
	}else{
		OUTPUT(i,j) = 1;
	}
	
}		
